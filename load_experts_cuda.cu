#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <hipcub/hipcub.hpp>
#include <thrust/device_vector.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

#define BLOCK_SIZE 128

/* 1. 把需要load的experts存到unloaded数组中，并使用reduction和prefix sum计算需要load的数量。 */
__global__ void load_experts_kernel(
    float *device_modules,
    long *experts_info,
    long *experts_list,
    long *selected_experts,
    int layer_id,
    int token_num,
    int topk,
    int device_num,
    int *unloaded,
    int *block_num,
    int single_sel_num,
    int num_bytes)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int pos_id = 0;

    typedef hipcub::BlockReduce<int, BLOCK_SIZE> BlockReduce;
    __shared__ typename BlockReduce::TempStorage temp_storage;
    int thread_data = 0;

    if (idx < single_sel_num)
    {
        int flag = 0;
        for (int k = 0; k < device_num; ++k)
        {
            if (selected_experts[idx] + layer_id * single_sel_num == experts_info[k])
            {
                pos_id = k;
                flag = 1;
                break;
            }
        }
        thread_data = (flag == 0);
        unloaded[idx] = selected_experts[idx] * (flag == 0) + (flag == 1) * (-1);

        experts_list[idx] = pos_id * num_bytes;
    }

    int aggregate = BlockReduce(temp_storage).Sum(thread_data);
    if (threadIdx.x == 0)
    {
        block_num[blockIdx.x] = aggregate;
    }
}

__global__ void prefix_sum(int *src, int *dst, int n)
{
    if (threadIdx.x == 0 && blockIdx.x == 0)
    {
        dst[0] = 0;
        for (int i = 1; i < n; i++)
        {
            dst[i] = dst[i - 1] + src[i - 1];
        }
    }
}

/* 2. 根据需要load的数量，从前到后选取experts_prefer_order中元素作为pos_id。最后再对experts_prefer_order进行排序。 */
__global__ void load_experts_list_kernel(
    float *device_modules,
    float *offloaded_modules,
    long *experts_info,
    long *experts_prefer_order,
    long *tmp_experts_prefer_order,
    long *experts_list,
    int *unloaded,
    int *unloaded_num,
    int layer_id,
    int single_sel_num,
    int device_num,
    int grid_size,
    int num_bytes)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    long pos_id = 0;

    int important_experts = device_num - unloaded_num[grid_size] + idx;
    int less_important_experts = idx + unloaded_num[grid_size];

    if (idx < unloaded_num[grid_size])
    {
        pos_id = experts_prefer_order[idx];
        int j = -1;
        for (int i = 0; i < idx + 1; i++)
        {
            j++;
            while (unloaded[j] == -1)
            {
                j++;
            }
        }
        device_modules[2 * pos_id] = offloaded_modules[2 * (unloaded[j] + layer_id * single_sel_num)];
        device_modules[2 * pos_id + 1] = offloaded_modules[2 * (unloaded[j] + layer_id * single_sel_num) + 1];
        experts_info[pos_id] = unloaded[j] + layer_id * single_sel_num;
        experts_list[j] = pos_id * num_bytes;
    }
    if (less_important_experts < device_num)
    {
        tmp_experts_prefer_order[less_important_experts] = experts_prefer_order[less_important_experts];
        experts_prefer_order[idx] = tmp_experts_prefer_order[less_important_experts];
    }
    if (important_experts < device_num)
    {
        experts_prefer_order[important_experts] = pos_id;
    }
}

void load_experts_cuda(
    float *device_modules,
    float *offloaded_modules,
    long *experts_info,
    long *selected_experts,
    long *experts_prefer_order,
    int layer_id,
    long *experts_list,
    int offloaded_num,
    int token_num,
    int topk,
    int device_num)
{
    int grid_size = (offloaded_num + BLOCK_SIZE - 1) / BLOCK_SIZE;
    int num_bytes = 2 * sizeof(device_modules[0]);
    dim3 grid(grid_size);
    dim3 block(BLOCK_SIZE);

    float *d_offloaded_modules = nullptr;
    int *d_unloaded = nullptr;
    int *unloaded_num = nullptr;
    int *block_num = nullptr;
    long *tmp_experts_prefer_order = nullptr;

    hipMalloc((void **)&d_offloaded_modules, 2 * offloaded_num * sizeof(offloaded_modules[0]));
    hipMalloc((void **)&d_unloaded, offloaded_num * sizeof(int));
    hipMalloc((void **)&unloaded_num, (grid_size + 1) * sizeof(int));
    hipMalloc((void **)&block_num, grid_size * sizeof(int));
    hipMalloc((void **)&tmp_experts_prefer_order, device_num * sizeof(long));

    hipMemcpy(d_offloaded_modules, offloaded_modules, 2 * offloaded_num * sizeof(offloaded_modules[0]), hipMemcpyHostToDevice);

    thrust::device_ptr<long> d_selected_experts(selected_experts);
    thrust::device_vector<long> d_vec_selected_experts(d_selected_experts, d_selected_experts + token_num*topk);
    thrust::sort(d_vec_selected_experts.begin(), d_vec_selected_experts.end());
    thrust::device_vector<long>::iterator new_end = thrust::unique(d_vec_selected_experts.begin(), d_vec_selected_experts.end());
    int single_sel_num = new_end - d_vec_selected_experts.begin();
    d_vec_selected_experts.resize(single_sel_num);

    load_experts_kernel<<<grid, block>>>(
        device_modules,
        experts_info,
        experts_list,
        selected_experts,
        layer_id,
        token_num,
        topk,
        device_num,
        d_unloaded,
        block_num,
        single_sel_num,
        num_bytes);

    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_storage, temp_storage_bytes,
        block_num, unloaded_num, grid_size + 1);

    // Allocate temporary storage
    hipMalloc(&d_temp_storage, temp_storage_bytes);

    // Run exclusive prefix sum
    hipcub::DeviceScan::ExclusiveSum(
        d_temp_storage, temp_storage_bytes,
        block_num, unloaded_num, grid_size + 1);


    //prefix_sum<<<1, 1>>>(block_num, unloaded_num, grid_size + 1);

    load_experts_list_kernel<<<grid, block>>>(
        device_modules,
        d_offloaded_modules,
        experts_info,
        experts_prefer_order,
        tmp_experts_prefer_order,
        experts_list,
        d_unloaded,
        unloaded_num,
        layer_id,
        single_sel_num,
        device_num,
        grid_size,
        num_bytes);

    hipFree(d_offloaded_modules);
    hipFree(d_unloaded);
    hipFree(unloaded_num);
    hipFree(block_num);
    hipFree(tmp_experts_prefer_order);
}

// int main(){
//     int device_num = 5;
//     int offloaded_num = 10;
//     int layer_id = 0;
//     int token_num = 1;
//     int topk = 4;
    
//     float *device_modules = nullptr;
//     float *offloaded_modules = nullptr;
//     long *experts_info = nullptr;
//     long *selected_experts = nullptr;
//     long *experts_prefer_order = nullptr;
//     long *experts_list = nullptr;

//     hipMalloc((void **)&device_modules, 2 * device_num * sizeof(float));
//     hipMalloc((void **)&offloaded_modules, 2 * offloaded_num * sizeof(float));
//     hipMalloc((void **)&experts_info, 2 * device_num * sizeof(long));
//     hipMalloc((void **)&selected_experts, 2 * topk * sizeof(long));
//     hipMalloc((void **)&experts_prefer_order, 2 * device_num * sizeof(long));
//     hipMalloc((void **)&experts_list, 2 * topk * sizeof(long));

//     load_experts_cuda(
//         device_modules,
//         offloaded_modules,
//         experts_info,
//         selected_experts,
//         experts_prefer_order,
//         layer_id,
//         experts_list,
//         offloaded_num,
//         token_num,
//         topk,
//         device_num);
// }
